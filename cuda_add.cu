#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stddef.h>
#include <stdlib.h>

__global__ void VecAdd(float* A, float* B, float* C, int  N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 10;
    size_t size = N * sizeof(float);

    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);
    h_A[0] = 2.2f;
    h_B[0] = 3.3f;

    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    printf("%f\n", h_C[0]);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    return 0;
}
